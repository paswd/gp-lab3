#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <vector>
//#include "../lib/cuPrintf.cu"

using namespace std;

const uint32_t BLOCK_DIM = 32;
const uint32_t MAX_CLASS_COUNT = 32;
//const uint32_t MAX_CLASS_ELEMENTS_COUNT = 524288;

namespace Pixel {
	//const uint32_t ELEMENTS_CNT = 4;
	const uint32_t RED = 8 * 0;
	const uint32_t GREEN = 8 * 1;
	const uint32_t BLUE = 8 * 2;
	const uint32_t ALPHA = 8 * 3;
}


/*
==========
STRUCTURES
==========
*/

struct Position {
	int32_t X;
	int32_t Y;
};
struct ModifiedPixel {
	float Red;
	float Green;
	float Blue;
};

texture<uint32_t, 2, hipReadModeElementType> OriginalImage;
//__constant__ Position class_map[MAX_CLASS_COUNT * MAX_CLASS_ELEMENTS_COUNT];
__constant__ uint8_t ClassCount[1];
__constant__ uint32_t ClassMapElementsCounts[MAX_CLASS_COUNT];
__constant__ uint32_t ClassMapElementsOffsets[MAX_CLASS_COUNT];
__constant__ ModifiedPixel ClassAVG[MAX_CLASS_COUNT];
texture<Position, 1, hipReadModeElementType> ClassMap;

/*
===========
DEVICE-HOST
===========
*/

/*__device__ double GetIntensity(Pixel pixel) {
	return (.3 * (double) pixel.Red) + (.59 * (double) pixel.Green) + (.11 * (double) pixel.Blue);
}*/



__host__ __device__ ModifiedPixel SetModifiedPixel() {
	ModifiedPixel res;
	res.Red = 0.;
	res.Green = 0.;
	res.Blue = 0.;
	return res;
}

__host__ __device__ ModifiedPixel SetModifiedPixel(float r, float g, float b) {
	ModifiedPixel res;
	res.Red = r;
	res.Green = g;
	res.Blue = b;
	return res;
}
__host__ __device__ ModifiedPixel SetModifiedPixel(uint8_t r, uint8_t g, uint8_t b) {
	ModifiedPixel res;
	res.Red = (float) r;
	res.Green = (float) g;
	res.Blue = (float) b;
	return res;
}

__device__ __host__ bool IsCorrectPos(Position pos, uint32_t height, uint32_t width) {
	return (pos.X >= 0 && pos.Y >= 0 && pos.X < (int32_t) width && pos.Y < (int32_t) height);
}

__device__ __host__ int32_t GetLinearizedPosition(Position pos, uint32_t height, uint32_t width) {
	return (IsCorrectPos(pos, height, width)) ? (pos.Y * (int32_t) width + pos.X) : -1;
}
__device__ __host__ uint32_t MakePixel(uint8_t red, uint8_t green, uint8_t blue, uint8_t alpha) {
	return ((uint32_t) red << Pixel::RED) + ((uint32_t) green << Pixel::GREEN) +
			((uint32_t) blue << Pixel::BLUE) + ((uint32_t) alpha << Pixel::ALPHA);
}

__device__ __host__ uint8_t GetPixelElement(uint32_t pixel, uint32_t element) {
	return (uint8_t) (pixel >> element) & 255;
}

__device__ __host__ uint32_t SetPixelElement(uint32_t pixel, uint32_t element, uint8_t value) {
	return (~(255 << element) & pixel) + (((uint32_t) value) << element);
	//cout << element << endl;
	//return 255 >> element;
}

__device__ __host__ void PixelSumm(ModifiedPixel &a, ModifiedPixel b) {
	//return ModifiedPixel(a.Red + b.Red, a.Green + b.Green, a.Blue + b.Blue);
	a.Red += b.Red;
	a.Green += b.Green;
	a.Blue += b.Blue;
}
__device__ __host__ void PixelDiff(ModifiedPixel &a, ModifiedPixel b) {
	//return ModifiedPixel(a.Red - b.Red, a.Green - b.Green, a.Blue - b.Blue);
	a.Red -= b.Red;
	a.Green -= b.Green;
	a.Blue -= b.Blue;
}
__device__ __host__ float PixelMult(ModifiedPixel a, ModifiedPixel b) {
	//return ModifiedPixel(a.Red * b.Red, a.Green * b.Green, a.Blue * b.Blue);
	return (a.Red * b.Red) + (a.Green * b.Green) + (a.Blue * b.Blue);
}
__device__ __host__ void PixelMultNum(ModifiedPixel &a, float num) {
	//return ModifiedPixel(a.Red * num, a.Green * num, a.Blue * num);
	a.Red *= num;
	a.Green *= num;
	a.Blue *= num;
}
__device__ __host__ ModifiedPixel PixelMinus(ModifiedPixel a) {
	return SetModifiedPixel(-a.Red, -a.Green, -a.Blue);
}
__device__ __host__ ModifiedPixel ConvertPixelToModified(uint32_t pixel) {
	return SetModifiedPixel(GetPixelElement(pixel, Pixel::RED), GetPixelElement(pixel, Pixel::GREEN),
		GetPixelElement(pixel, Pixel::BLUE));
}
__device__ __host__ uint32_t ConvertPixelFromModified(ModifiedPixel mod_pixel) {
	return MakePixel((uint8_t) mod_pixel.Red, (uint8_t) mod_pixel.Green, (uint8_t) mod_pixel.Blue, 0);
}

/*
======
DEVICE
======
*/

__device__ float GetMinDist(Position pos, uint8_t j) {
	ModifiedPixel px_bas = ConvertPixelToModified(tex2D(OriginalImage, pos.X, pos.Y));
	//cuPrintf("%f:%f:%f\n", px_bas.Red, px_bas.Green, px_bas.Blue);
	//cuPrintf("%f:%f:%f\n", ClassAVG[j].Red, ClassAVG[j].Green, ClassAVG[j].Blue);

	PixelDiff(px_bas, ClassAVG[j]);
	//cuPrintf("%f:%f:%f\n", px_bas.Red, px_bas.Green, px_bas.Blue);
	return PixelMult(px_bas, PixelMinus(px_bas));
}

__device__ void SetClass(Position pos, uint32_t *map_out, uint32_t height, uint32_t width) {
	uint8_t class_number = 0;
	float max_val = 0.;
	uint8_t is_defined = 0;
	//cuPrintf("ClassCount = %d\n", (uint32_t) ClassCount[0]);

	for (uint8_t j = 0; j < ClassCount[0]; j++) {
		if (!is_defined) {
			max_val = GetMinDist(pos, j);
			class_number = j;
			is_defined = 1;
			continue;
		}

		//cuPrintf("j: %d\nMax: %f\nCurr: %f\n", j, max_val, GetMinDist(pos, j));

		if (GetMinDist(pos, j) > max_val) {
			max_val = GetMinDist(pos, j);
			class_number = j;
		}
	}

	//Class calculating

	map_out[GetLinearizedPosition(pos, height, width)] = tex2D(OriginalImage, pos.X, pos.Y);
	map_out[GetLinearizedPosition(pos, height, width)] = SetPixelElement(
		map_out[GetLinearizedPosition(pos, height, width)], Pixel::ALPHA, class_number);
}

/*
======
GLOBAL
======
*/

__global__ void Classificator(uint32_t height, uint32_t width, uint32_t *map_out) {

	Position start, offset;
	start.X = blockIdx.x * blockDim.x + threadIdx.x;
	start.Y = blockIdx.y * blockDim.y + threadIdx.y;

	offset.X = gridDim.x * blockDim.x;
	offset.Y = gridDim.y * blockDim.y;

	Position pos;
	for (pos.X = start.X; pos.X < width; pos.X += offset.X) {
		for (pos.Y = start.Y; pos.Y < height; pos.Y += offset.Y) {
			if (pos.X < width && pos.Y < height) {
				//cuPrintf("\n%d:%d\n", pos.X, pos.Y);
				SetClass(pos, map_out, height, width);
			}
		}
	}
}

/*
====
HOST
====
*/

__host__ void InitPixelMap(uint32_t **pixel, uint32_t height, uint32_t width) {
	*pixel = new uint32_t[height * width];
}

__host__ void DestroyPixelMap(uint32_t **pixel) {
	delete [] (*pixel);
	*pixel = NULL;
}	

__host__ void ReadImageFromFile(uint32_t **pixel, uint32_t *height, uint32_t *width,
		string filename) {
	FILE *file = fopen(filename.c_str(), "rb");
	uint32_t sizes[2];
	fread(sizes, sizeof(uint32_t), 2, file);
	*width = sizes[0];
	*height = sizes[1];

	uint32_t size = (*height) * (*width);

	InitPixelMap(pixel, *height, *width);
	fread(*pixel, sizeof(uint32_t), size, file);
	fclose(file);
}

__host__ void WriteImageToFile(uint32_t *pixel, uint32_t height, uint32_t width, string filename) {
	FILE *file = fopen(filename.c_str(), "wb");
	uint32_t sizes[2] = {width, height};
	fwrite(sizes, sizeof(uint32_t), 2, file);

	uint32_t size = height * width;
	fwrite(pixel, sizeof(uint32_t), size, file);
	fclose(file);
}

__host__ void FileGeneratorTest() {
	uint32_t *pixel;
	uint32_t height = 3;
	uint32_t width = 3;
	InitPixelMap(&pixel, height, width);

	string filename = "in.data";
	pixel[0] = MakePixel(162, 223, 76, 0);
	pixel[1] = MakePixel(247, 201, 254, 0);
	pixel[2] = MakePixel(158, 216, 69, 0);

	pixel[3] = MakePixel(180, 232, 83, 0);
	pixel[4] = MakePixel(153, 209, 77, 0);
	pixel[5] = MakePixel(146, 221, 86, 0);

	pixel[6] = MakePixel(169, 224, 76, 0);
	pixel[7] = MakePixel(247, 209, 250, 0);
	pixel[8] = MakePixel(212, 208, 233, 0);

	WriteImageToFile(pixel, height, width, filename);
	DestroyPixelMap(&pixel);
}
__host__ uint32_t GetRandomPixel() {
	return MakePixel(rand() % 256, rand() % 256, rand() % 256, 0);
}
__host__ void FileGenerator(string filename) {
	uint32_t *pixel;
	uint32_t height = 400;
	uint32_t width = 640;
	InitPixelMap(&pixel, height, width);

	for (uint32_t i = 0; i < height; i++) {
		for (uint32_t j = 0; j < width; j++) {
			pixel[i * width + j] = GetRandomPixel();
		}
	}
	WriteImageToFile(pixel, height, width, filename);
	DestroyPixelMap(&pixel);
}
__host__ void FileGeneratorBig(uint32_t height, uint32_t width, string filename) {
	uint32_t *pixel;
	InitPixelMap(&pixel, height, width);

	for (uint32_t i = 0; i < height; i++) {
		for (uint32_t j = 0; j < width; j++) {
			uint8_t curr;
			if (i == 0 || j == 0 || i == height - 1 || j == width - 1) {
				curr = 1;
			} else {
				curr = 3;
			}
			pixel[i * width + j] = MakePixel(curr, curr, curr, 0);
		}
	}

	WriteImageToFile(pixel, height, width, filename);
	DestroyPixelMap(&pixel);
}

__host__ ModifiedPixel GetAVG(uint8_t j, uint32_t *map_in, uint32_t height, uint32_t width,
		vector <Position> &class_elements, uint32_t *class_elements_counts,
		uint32_t *class_elements_offsets) {
	ModifiedPixel res = SetModifiedPixel();
	//printf("%f:%f:%f\n", res.Red, res.Green, res.Blue);


	for (uint32_t i = class_elements_offsets[j];
			i < class_elements_offsets[j] + class_elements_counts[j]; i++) {
		//PixelSumm(res, ConvertPixelToModified(tex2D(OriginalImage, tex1D(ClassMap, i).X, tex1D(ClassMap, i).Y)));
		PixelSumm(res, ConvertPixelToModified(map_in[GetLinearizedPosition(class_elements[i], height, width)]));
	
	}
	//printf("%f:%f:%f\n", res.Red, res.Green, res.Blue);
	PixelMultNum(res, 1./((float) class_elements_counts[j]));
	//printf("%f:%f:%f\n", res.Red, res.Green, res.Blue);
	return res;
}

__host__ int main(void) {
	//cout << "INIT" << endl;
	srand(time(NULL));
	//FileGeneratorBig(100, 100, "inbig.data");
	//FileGenerator("inrand.data");
	//FileGeneratorTest();

	string file_in, file_out;
	cin >> file_in >> file_out;
	//cout << "FILE INIT COMPLETED" << endl;

	//FileGenerator();
	uint32_t *pixel_in;
	uint32_t *pixel_out;
	uint32_t height, width;

	ReadImageFromFile(&pixel_in, &height, &width, file_in);
	//cout << "READ IMAGE COMPLETED" << endl;

	InitPixelMap(&pixel_out, height, width);

	uint32_t class_elements_counts[MAX_CLASS_COUNT];
	uint32_t class_elements_offsets[MAX_CLASS_COUNT];
	ModifiedPixel class_elements_avg[MAX_CLASS_COUNT];

	vector <Position> class_elements(0);
	uint32_t cnt = 0;

	//cout << "INPUT CLASSES" << endl;

	uint8_t class_count;
	uint32_t tmp;
	cin >> tmp;
	class_count = (uint8_t) tmp;
	//cout << class_count << endl;
	for (uint8_t i = 0; i < class_count; i++) {
		//cout << "CL_CNT = " << class_count << endl;
		//cout << "i = " << (uint32_t) i << endl;
		//cout << (uint32_t) i << " < " << class_count << " = ";
		//cout << (i < class_count ? "true" : "false") << endl;
		cin >> class_elements_counts[i];
		//cout << class_elements_counts[i] << endl;
		class_elements_offsets[i] = cnt;
		cnt += class_elements_counts[i];
		for (uint32_t j = 0; j < class_elements_counts[i]; j++) {
			//cout << "\t" << (uint32_t) j << endl;
			Position tmp_pos;
			cin >> tmp_pos.X >> tmp_pos.Y;
			class_elements.push_back(tmp_pos);
		}
		class_elements_avg[i] = GetAVG(i, pixel_in, height, width, class_elements,
			class_elements_counts, class_elements_offsets);
		//printf("%f:%f:%f\n", class_elements_avg[i].Red, class_elements_avg[i].Green, class_elements_avg[i].Blue);

	}
	//cout << "INPUT END" << endl;

	uint32_t *cuda_pixel_out;

	//Texture init begin
	hipArray *cuda_pixel_in;
	hipChannelFormatDesc ch1 = hipCreateChannelDesc<uint32_t>();
	hipMallocArray(&cuda_pixel_in, &ch1, width, height);
	hipMemcpyToArray(cuda_pixel_in, 0, 0, pixel_in, sizeof(uint32_t) * height * width, hipMemcpyHostToDevice);
	
	OriginalImage.addressMode[0] = hipAddressModeClamp;
	OriginalImage.addressMode[1] = hipAddressModeClamp;

	OriginalImage.channelDesc = ch1;
	OriginalImage.filterMode = hipFilterModePoint;
	OriginalImage.normalized = false;
	hipBindTextureToArray(OriginalImage, cuda_pixel_in, ch1);
	//Texture init end

	//Texture init begin
	hipArray *cuda_classmap;
	hipChannelFormatDesc ch2 = hipCreateChannelDesc<Position>();
	hipMallocArray(&cuda_classmap, &ch2, width, height);
	hipMemcpyToArray(cuda_classmap, 0, 0, pixel_in, sizeof(uint32_t) * height * width, hipMemcpyHostToDevice);
	
	ClassMap.addressMode[0] = hipAddressModeClamp;
	ClassMap.addressMode[1] = hipAddressModeClamp;

	ClassMap.channelDesc = ch2;
	ClassMap.filterMode = hipFilterModePoint;
	ClassMap.normalized = false;
	hipBindTextureToArray(ClassMap, cuda_classmap, ch2);
	//Texture init end

	hipMemcpyToSymbol(HIP_SYMBOL(ClassMapElementsCounts), class_elements_counts, sizeof(uint32_t) * class_count);
	hipMemcpyToSymbol(HIP_SYMBOL(ClassMapElementsOffsets), class_elements_offsets, sizeof(uint32_t) * class_count);
	hipMemcpyToSymbol(HIP_SYMBOL(ClassCount), &class_count, sizeof(uint8_t));
	hipMemcpyToSymbol(HIP_SYMBOL(ClassAVG), class_elements_avg, sizeof(ModifiedPixel) * class_count);
	hipMalloc((void**) &cuda_pixel_out, sizeof(uint32_t) * width * height);

	dim3 threads_per_block(width, height);
	dim3 blocks_per_grid(1, 1);

	if (height * width > BLOCK_DIM * BLOCK_DIM){
		threads_per_block.x = BLOCK_DIM;
		threads_per_block.y = BLOCK_DIM;
		blocks_per_grid.x = ceil((double) (width) / (double)(threads_per_block.x));
		blocks_per_grid.y = ceil((double) (height) / (double)(threads_per_block.y));
	}

	//cudaPrintfInit();
	Classificator<<<blocks_per_grid, threads_per_block>>>(height, width, cuda_pixel_out);
	//cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(pixel_out, cuda_pixel_out, sizeof(uint32_t) * width * height, hipMemcpyDeviceToHost);

	hipEventDestroy(syncEvent);

	hipUnbindTexture(OriginalImage);
	hipFreeArray(cuda_pixel_in);
	hipFree(cuda_pixel_out);

	WriteImageToFile(pixel_out, height, width, file_out);

	DestroyPixelMap(&pixel_in);
	DestroyPixelMap(&pixel_out);

	return 0;
}

